#include "hip/hip_runtime.h"
// cublas
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// cusparselt
#include <iostream>
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <cusparseLt.h>       // cusparseLt header
#include <cstdio>             // printf
#include <cstdlib>            // std::rand


#include <mma.h>
using namespace nvcuda;

#include "../util/utility.h"

#include <mma.h>
using namespace nvcuda;

#define TEST_TIMES 10

#if TEST_TIMES > 1
    float alpha = 1.0, beta_ = 1.0;
#else
    float alpha = 1.0, beta_ = 0.0;
#endif

__global__ void convertFp32ToFp16 (__half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// reference: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/tensor-cores/simpleTensorCoreGEMM.cu
__global__ void bsr_wmma(half *a, half *b, float *c, int m, int n, int k, float alpha, float beta,
                         const int *__restrict__ rowptr, const int *__restrict__ colind)
{
   // Leading dimensions. Packed with no transpositions.
   int lda = m;
   int ldb = k;
   int ldc = m;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // infer aRow and bCol from warp
   int aRow = warpM * WMMA_M;
   int bCol = warpN * WMMA_N;

   // loop over colind
   for (int i = rowptr[warpM]; i < rowptr[warpM+1]; i += 1) {

      int aCol = colind[i] * WMMA_K;
      int bRow = colind[i] * WMMA_K;

      // Bounds checking
      if (aRow < m && aCol < k && bRow < k && bCol < n) {

         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + i * WMMA_M * WMMA_K, WMMA_M);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < m && cCol < n) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}

// m16n16k16
__global__ void bsr_wmma_half_half_half(__half *a, __half *b, __half *c, int m, int n, int k, __half alpha, __half beta,
                                       const int *__restrict__ rowptr, const int *__restrict__ colind)
{
   // Leading dimensions. Packed with no transpositions.
   int lda = m;
   int ldb = k;
   int ldc = m;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
   wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

   wmma::fill_fragment(acc_frag, static_cast<__half>(0.0f));

   // infer aRow and bCol from warp
   int aRow = warpM * WMMA_M;
   int bCol = warpN * WMMA_N;

   // loop over colind
   for (int i = rowptr[warpM]; i < rowptr[warpM+1]; i += 1) {

      int aCol = colind[i] * WMMA_K;
      int bRow = colind[i] * WMMA_K;

      // Bounds checking
      if (aRow < m && aCol < k && bRow < k && bCol < n) {

         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + i * WMMA_M * WMMA_K, WMMA_M);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cRow = warpM * WMMA_M;
   int cCol = warpN * WMMA_N;

   if (cRow < m && cCol < n) {
      wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

#pragma unroll
      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
   }
}

double evalCuBLASHGemm(__half *hA, __half *hB, __half *hC, 
                       int m, int n, int k)
{
   // Because CUBLAS uses column major, C^T = B^T * A^T.
   bool trans_A = false;
   bool trans_B = true;
   hipblasOperation_t cublas_trans_A = trans_A?HIPBLAS_OP_T:HIPBLAS_OP_N;
   hipblasOperation_t cublas_trans_B = trans_B?HIPBLAS_OP_T:HIPBLAS_OP_N;

   __half* hfA = NULL; 
   hipMalloc(&hfA, m*k*sizeof(__half));
   hipMemcpy(hfA, hA, m*k*sizeof(__half), hipMemcpyHostToDevice);

   __half* hfB = NULL;
   hipMalloc(&hfB, k*n*sizeof(__half));
   hipMemcpy(hfB, hB, k*n*sizeof(__half), hipMemcpyHostToDevice);

   __half* hfC = NULL;
   hipMalloc(&hfC, m*n*sizeof(__half));
   hipMemcpy(hfC, hC, m*n*sizeof(__half), hipMemcpyHostToDevice);

   hipblasHandle_t handle;
   hipblasCreate(&handle);

   // convert alpha, beta to half
   __half hf_alpha = __float2half(alpha);
   __half hf_beta = __float2half(beta_);

   //----------------------- 
   // warm up
   hipblasHgemm(handle, cublas_trans_B, cublas_trans_A, n, m, k,
   &hf_alpha, hfB, n, hfA, k, &hf_beta, hfC, n);
   // hipblasHgemm(handle, cublas_trans_A, cublas_trans_B, M, N, K,
   // &hf_alpha, hfA, M, hfB, K, &hf_beta, hfC, M);

   GpuTimer cublas_timer;
   cublas_timer.Start();
   for (int i=0; i<TEST_TIMES; i++)
   {
      hipblasHgemm(handle, cublas_trans_B, cublas_trans_A, n, m, k,
                  &hf_alpha, hfB, n, hfA, k, &hf_beta, hfC, n);
      // hipblasHgemm(handle, cublas_trans_A, cublas_trans_B, M, N, K,
      // &hf_alpha, hfA, M, hfB, K, &hf_beta, hfC, M);
   }
   cublas_timer.Stop();
   double cublas_time = cublas_timer.ElapsedMillis()/TEST_TIMES;
   //----------------------- 
   hipMemcpy(hC, hfC, m * n *sizeof(__half), hipMemcpyDeviceToHost);
   __half *hC_trans = (__half*)malloc(m * n * sizeof(__half));
   transpose(hC_trans, hC, m, n);
   hipMemcpy(hC, hC_trans, m * n * sizeof(__half), hipMemcpyHostToHost);
   free(hC_trans);

   // hipFree(hfA);
   // hipFree(hfB);
   // hipFree(hfC);

   return cublas_time;
}

double evaluCuBLASGemmex(__half *hA, __half *hB, __half *hC, 
                        int m, int n, int k)
{
   hipblasHandle_t cublasH = NULL;
   // hipStream_t stream = NULL;

   const int lda = m;
   const int ldb = k;
   const int ldc = m;
   /*
   *   A = | 1.0 | 2.0 |
   *       | 3.0 | 4.0 |
   *
   *   B = | 5.0 | 6.0 |
   *       | 7.0 | 8.0 |
   */

   // const std::vector<data_type> A = {1.0, 3.0, 2.0, 4.0};
   // const std::vector<data_type> B = {5.0, 7.0, 6.0, 8.0};
   // std::vector<data_type> C(m * n);
   // const data_type alpha = 1.0;
   // const data_type beta = 0.0;

   __half *d_A = nullptr;
   __half *d_B = nullptr;
   __half *d_C = nullptr;

   hipblasOperation_t transa = HIPBLAS_OP_N;
   hipblasOperation_t transb = HIPBLAS_OP_N;

   // printf("A\n");
   // print_matrix(m, k, A.data(), lda);
   // printf("=====\n");

   // printf("B\n");
   // print_matrix(k, n, B.data(), ldb);
   // printf("=====\n");

   __half hf_alpha = __float2half(alpha);
   __half hf_beta = __float2half(beta_);

   /* step 1: create cublas handle, bind a stream */
   hipblasCreate(&cublasH);

   // CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
   // CUBLAS_CHECK(hipblasSetStream(cublasH, stream));

   /* step 2: copy data to device */
   CHECK_CUDA( hipMalloc((void **)&d_A, sizeof(__half) * m * k) )
   CHECK_CUDA( hipMalloc((void **)&d_B, sizeof(__half) * k * n) )
   CHECK_CUDA( hipMalloc((void **)&d_C, sizeof(__half) * m * n) )

   __half *hA_trans = (__half*)malloc(m * k * sizeof(__half));
   transpose(hA_trans, hA, m, k);

   CHECK_CUDA( hipMemcpy(d_A, hA_trans, sizeof(__half) * m * k, hipMemcpyHostToDevice) )
   CHECK_CUDA( hipMemcpy(d_B, hB, sizeof(__half) * k * n, hipMemcpyHostToDevice) )

   /* step 3: compute */
   hipblasGemmEx(cublasH, transa, transb, m, n, k, &hf_alpha, d_A, HIP_R_16F, lda, d_B,
   HIP_R_16F, ldb, &hf_beta, d_C, HIP_R_16F, ldc,
   HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

   GpuTimer cublas_timer;
   cublas_timer.Start();
   for (int i=0; i<TEST_TIMES; i++)
   {
      hipblasGemmEx(cublasH, transa, transb, m, n, k, &hf_alpha, d_A, HIP_R_16F, lda, d_B,
      HIP_R_16F, ldb, &hf_beta, d_C, HIP_R_16F, ldc,
      HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
   }
   cublas_timer.Stop();
   double cublas_time = cublas_timer.ElapsedMillis()/TEST_TIMES;

   /* step 4: copy data to host */
   CHECK_CUDA( hipMemcpy(hC, d_C, sizeof(__half) * m * n, hipMemcpyDeviceToHost) )

   // CUDA_CHECK(hipStreamSynchronize(stream));

   /*
   *   C = | 19.0 | 22.0 |
   *       | 43.0 | 50.0 |
   */

   // printf("C\n");
   // print_matrix(m, n, C.data(), ldc);
   // printf("=====\n");

   /* free resources */
   // CUDA_CHECK(hipFree(d_A));
   // CUDA_CHECK(hipFree(d_B));
   // CUDA_CHECK(hipFree(d_C));

   hipblasDestroy(cublasH);

   // CUDA_CHECK(hipStreamDestroy(stream));

   // CUDA_CHECK(hipDeviceReset());
   return cublas_time;
}

double evalCuSPARSELtMatmul(__half *hA, __half *hB, __half *hC, int m, int n, int k)
{
   // Host problem definition, row-major order
   // bigger sizes may require dynamic allocations
   auto          order        = HIPSPARSE_ORDER_ROW;
   auto          opA          = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   auto          opB          = HIPSPARSE_OPERATION_NON_TRANSPOSE;
   auto          type         = HIP_R_16F;
   auto          compute_type = CUSPARSE_COMPUTE_16F;

   bool     is_rowmajor    = (order == HIPSPARSE_ORDER_ROW);
   bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
   bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
   auto     num_A_rows     = (isA_transposed) ? k : m;
   auto     num_A_cols     = (isA_transposed) ? m : k;
   auto     num_B_rows     = (isB_transposed) ? n : k;
   auto     num_B_cols     = (isB_transposed) ? k : n;
   auto     num_C_rows     = m;
   auto     num_C_cols     = n;
   unsigned alignment      = 16;
   auto     lda            = (is_rowmajor) ? num_A_cols : num_A_rows;
   auto     ldb            = (is_rowmajor) ? num_B_cols : num_B_rows;
   auto     ldc            = (is_rowmajor) ? num_C_cols : num_C_rows;
   auto     A_height       = (is_rowmajor) ? num_A_rows : num_A_cols;
   auto     B_height       = (is_rowmajor) ? num_B_rows : num_B_cols;
   auto     C_height       = (is_rowmajor) ? num_C_rows : num_C_cols;
   auto     A_size         = A_height * lda * sizeof(__half);
   auto     B_size         = B_height * ldb * sizeof(__half);
   auto     C_size         = C_height * ldc * sizeof(__half);

   //--------------------------------------------------------------------------
   // Device memory management
   __half *dA, *dB, *dC, *dD, *dA_compressed;
   int    *d_valid;
   CHECK_CUDA( hipMalloc((void**) &dA, A_size) )
   CHECK_CUDA( hipMalloc((void**) &dB, B_size) )
   CHECK_CUDA( hipMalloc((void**) &dC, C_size) )
   CHECK_CUDA( hipMalloc((void**) &d_valid, sizeof(int)) )
   dD = dC;

   CHECK_CUDA( hipMemcpy(dA, hA, A_size, hipMemcpyHostToDevice) )
   CHECK_CUDA( hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice) )
   CHECK_CUDA( hipMemcpy(dC, hC, C_size, hipMemcpyHostToDevice) )
   //--------------------------------------------------------------------------
   cusparseLtHandle_t             handle;
   cusparseLtMatDescriptor_t      matA, matB, matC;
   cusparseLtMatmulDescriptor_t   matmul;
   cusparseLtMatmulAlgSelection_t alg_sel;
   cusparseLtMatmulPlan_t         plan;
   hipStream_t                   stream = nullptr;
   CHECK_CUSPARSE( cusparseLtInit(&handle) )

   // matrix descriptor initialization
   CHECK_CUSPARSE( cusparseLtStructuredDescriptorInit(
                                          &handle, &matA, num_A_rows,
                                          num_A_cols, lda, alignment,
                                          type, order,
                                          CUSPARSELT_SPARSITY_50_PERCENT) )
   CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                          &handle, &matB, num_B_rows,
                                          num_B_cols, ldb, alignment,
                                          type, order) )
   CHECK_CUSPARSE( cusparseLtDenseDescriptorInit(
                                          &handle, &matC, num_C_rows,
                                          num_C_cols, ldc, alignment,
                                          type, order) )
   // matmul, algorithm selection, and plan initialization
   CHECK_CUSPARSE( cusparseLtMatmulDescriptorInit(
                                          &handle, &matmul, opA, opB,
                                          &matA, &matB, &matC, &matC,
                                          compute_type) )
   CHECK_CUSPARSE( cusparseLtMatmulAlgSelectionInit(
                                          &handle, &alg_sel, &matmul,
                                          CUSPARSELT_MATMUL_ALG_DEFAULT) )
   CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel))

   //--------------------------------------------------------------------------
   // Prune the A matrix (in-place) and check the correctness
   CHECK_CUSPARSE( cusparseLtSpMMAPrune(&handle, &matmul, dA, dA,
                                       CUSPARSELT_PRUNE_SPMMA_TILE, stream) )
   CHECK_CUSPARSE( cusparseLtSpMMAPruneCheck(&handle, &matmul, dA,
                                             d_valid, stream) )
   int is_valid;
   CHECK_CUDA( hipMemcpyAsync(&is_valid, d_valid, sizeof(int),
                              hipMemcpyDeviceToHost, stream) )
   CHECK_CUDA( hipStreamSynchronize(stream) )
   if (is_valid != 0) {
      // std::printf("!!!! The matrix has been pruned in a wrong way. "
      //             "cusparseLtMatmul will not provide correct results\n");
      return EXIT_FAILURE;
   }

   //--------------------------------------------------------------------------
   // Compress the A matrix
   size_t compressed_size, compressed_buffer_size;
   void*  dA_compressedBuffer;
   CHECK_CUSPARSE( cusparseLtSpMMACompressedSize(&handle, &plan,
                                                &compressed_size,
                                                &compressed_buffer_size) )
   CHECK_CUDA( hipMalloc((void**) &dA_compressed, compressed_size) )
   CHECK_CUDA( hipMalloc((void**) &dA_compressedBuffer,
                        compressed_buffer_size) )

   CHECK_CUSPARSE( cusparseLtSpMMACompress(&handle, &plan, dA, dA_compressed,
                                          dA_compressedBuffer,stream) )
   //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
   // Search the best kernel
   int           num_streams = 0;
   hipStream_t* streams     = nullptr;

   CHECK_CUSPARSE( cusparseLtMatmulSearch(&handle, &plan, &alpha,
                                          dA_compressed, dB, &beta_,
                                          dC, dD, nullptr,
                                          streams, num_streams) )
                                          
   // otherwise, it is possible to set it directly:
   int alg = 0;
   CHECK_CUSPARSE( cusparseLtMatmulAlgSetAttribute(&handle, &alg_sel,
                                                   CUSPARSELT_MATMUL_ALG_CONFIG_ID,
                                                   &alg, sizeof(alg)))
   //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
   size_t workspace_size;
   CHECK_CUSPARSE( cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel))

   CHECK_CUSPARSE( cusparseLtMatmulGetWorkspace(&handle, &plan,
                                                &workspace_size))
   void* d_workspace;
   CHECK_CUDA( hipMalloc((void**) &d_workspace, workspace_size) )


   // ===========================================================
   // warm up
   cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB,
                  &beta_, dC, dD, d_workspace, streams,
                  num_streams);


   float milliseconds = 0;
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start);
   for (int i=0; i<TEST_TIMES; i++)
   {
      // Perform the matrix multiplication
      cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB,
                        &beta_, dC, dD, d_workspace, streams,
                        num_streams);
   }
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   milliseconds = 0;
   hipEventElapsedTime(&milliseconds,start,stop);
   double cusparselt_time = (milliseconds)/double(TEST_TIMES);
   // ===========================================================
   hipMemcpy(hC, dC, M*N*sizeof(__half), hipMemcpyDeviceToHost);

   //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
   // destroy plan and handle
   CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matA) )
   CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matB) )
   CHECK_CUSPARSE( cusparseLtMatDescriptorDestroy(&matC) )
   CHECK_CUSPARSE( cusparseLtMatmulPlanDestroy(&plan) )
   CHECK_CUSPARSE( cusparseLtDestroy(&handle) )

   //--------------------------------------------------------------------------
   // device memory deallocation
   CHECK_CUDA( hipFree(dA_compressed) )
   // CHECK_CUDA( hipFree(dA) )
   // CHECK_CUDA( hipFree(dB) )
   // CHECK_CUDA( hipFree(dC) )
   CHECK_CUDA( hipFree(d_valid) )
   CHECK_CUDA( hipFree(d_workspace) )
   CHECK_CUDA( hipFree(dA_compressedBuffer) )

   return cusparselt_time;
}


double evalCuSPARSESpMMBlockedell(int *ell_columns, __half *ell_values, int ell_width,
                                 __half *hB, __half *hC,
                                 int m, int n, int k, int block_dim=16)
{
   hipsparseSpMMAlg_t alg = HIPSPARSE_SPMM_ALG_DEFAULT; // HIPSPARSE_SPMM_BLOCKED_ELL_ALG1;

   // Host problem definition
   int   A_num_rows      = m;
   int   A_num_cols      = k;
   int   A_ell_blocksize = block_dim;
   int   A_ell_cols      = ell_width;
   int   A_num_blocks    = A_ell_cols * A_num_rows /
                        (A_ell_blocksize * A_ell_blocksize);
   int   B_num_rows      = A_num_cols;
   int   B_num_cols      = n;
   int   ldb             = B_num_rows;
   int   ldc             = A_num_rows;
   int   B_size          = ldb * B_num_cols;
   int   C_size          = ldc * B_num_cols;
   // int   *hA_columns     = h_ell_columns;
   // __half *hA_values     = h_ell_values;

   // Device memory management
   int    *dA_columns;
   __half *dA_values, *dB, *dC;
   dA_columns = ell_columns;
   dA_values = ell_values;

   // CHECK_CUDA( hipMalloc((void**) &dA_columns, A_num_blocks * sizeof(int)) )
   // CHECK_CUDA( hipMalloc((void**) &dA_values,
   //                               A_ell_cols * A_num_rows * sizeof(__half)) )
   // CHECK_CUDA( hipMemcpy(dA_columns, ell_columns,
   //                      A_num_blocks * sizeof(int),
   //                      hipMemcpyDeviceToDevice) )
   // CHECK_CUDA( hipMemcpy(dA_values, ell_values,
   //                      A_ell_cols * A_num_rows * sizeof(__half),
   //                      hipMemcpyDeviceToDevice) )

   CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(__half)) )
   CHECK_CUDA( hipMalloc((void**) &dC, C_size * sizeof(__half)) )
   CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(__half),
                           hipMemcpyHostToDevice) )
   CHECK_CUDA( hipMemcpy(dC, hC, C_size * sizeof(__half),
                           hipMemcpyHostToDevice) )

   //--------------------------------------------------------------------------
   // CUSPARSE APIs
   hipsparseHandle_t     bhandle = NULL;
   hipsparseSpMatDescr_t bmatA;
   hipsparseDnMatDescr_t bmatB, bmatC;
   void*                bdBuffer    = NULL;
   size_t               bbufferSize = 0;
   CHECK_CUSPARSE( hipsparseCreate(&bhandle) )

   // Create sparse matrix A in blocked ELL format
   CHECK_CUSPARSE( hipsparseCreateBlockedEll(&bmatA,
                                             A_num_rows, A_num_cols, A_ell_blocksize,
                                             A_ell_cols, dA_columns, dA_values,
                                             HIPSPARSE_INDEX_32I,
                                             HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F) )
   // Create dense matrix B
   CHECK_CUSPARSE( hipsparseCreateDnMat(&bmatB, A_num_cols, B_num_cols, ldb, dB,
                                       HIP_R_16F, HIPSPARSE_ORDER_COL) )
   // Create dense matrix C
   CHECK_CUSPARSE( hipsparseCreateDnMat(&bmatC, A_num_rows, B_num_cols, ldc, dC,
                                       HIP_R_16F, HIPSPARSE_ORDER_COL) )
   
   // allocate an external buffer if needed
   __half hf_alpha = __float2half(alpha);
   __half hf_beta = __float2half(beta_);

   CHECK_CUSPARSE( hipsparseSpMM_bufferSize(bhandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &hf_alpha, bmatA, bmatB, &hf_beta, bmatC, HIP_R_16F,
                                           alg, &bbufferSize) )
   CHECK_CUDA( hipMalloc(&bdBuffer, bbufferSize) )

   // execute SpMM
   // warm-up
   CHECK_CUSPARSE( hipsparseSpMM(bhandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &hf_alpha, bmatA, bmatB, &hf_beta, bmatC, HIP_R_16F,
                                 alg, bdBuffer) )

   GpuTimer cusparse_timer;
   cusparse_timer.Start();
   for (int i=0; i<TEST_TIMES; i++)
   {
      CHECK_CUSPARSE( hipsparseSpMM(bhandle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &hf_alpha, bmatA, bmatB, &hf_beta, bmatC, HIP_R_16F,
                                    alg, bdBuffer) )
   }
   cusparse_timer.Stop();
   double cusparse_time = cusparse_timer.ElapsedMillis()/TEST_TIMES;

   // destroy matrix/vector descriptors
   CHECK_CUSPARSE( hipsparseDestroySpMat(bmatA) )
   CHECK_CUSPARSE( hipsparseDestroyDnMat(bmatB) )
   CHECK_CUSPARSE( hipsparseDestroyDnMat(bmatC) )
   CHECK_CUSPARSE( hipsparseDestroy(bhandle) )

   //--------------------------------------------------------------------------
   // device result check
   CHECK_CUDA( hipMemcpy(hC, dC, C_size * sizeof(__half), hipMemcpyDeviceToHost) )

   // device memory deallocation
   CHECK_CUDA( hipFree(bdBuffer) )
   CHECK_CUDA( hipFree(dA_columns) )
   CHECK_CUDA( hipFree(dA_values) )
   // CHECK_CUDA( hipFree(dB) )
   // CHECK_CUDA( hipFree(dC) )

   return cusparse_time;
}

double evalCustomBsrwmma(int *bsrRowPtr, int *bsrColInd, __half *hbsrVal,
                        __half *hB, __half *hC,
                        int m, int n, int k, int block_dim=16)
{
   __half *dB, *dC;
   hipMalloc((void**)&dB, k * n * sizeof(__half));
   hipMalloc((void**)&dC, k * n * sizeof(__half));
   hipMemcpy(dB, hB, k * n * sizeof(__half), hipMemcpyHostToDevice);
   hipMemcpy(dC, hC, m * n * sizeof(__half), hipMemcpyHostToDevice);

   __half hf_alpha = __float2half(alpha);
   __half hf_beta = __float2half(beta_);

   // blockDim.x must be a multple of warpSize
   // 128x4 means we have 16 (4x4) warps and a block computes a 64x64 output tile
   dim3 gridDim;
   dim3 blockDim;

   blockDim.x = 128;
   blockDim.y = 4;
   gridDim.x = (m + (16 * blockDim.x / 32 - 1)) / (16 * blockDim.x / 32);
   gridDim.y = (n + 16 * blockDim.y - 1) / (16 * blockDim.y);

   // ------
   // warm up
   bsr_wmma_half_half_half<<<gridDim, blockDim>>>(hbsrVal, dB, dC, 
                                                   m, n, k, 
                                                   hf_alpha, hf_beta,
                                                   bsrRowPtr, bsrColInd);

   GpuTimer bsrwmma_timer;
   bsrwmma_timer.Start();
   for (int i = 0; i < TEST_TIMES; i++)
   {
      bsr_wmma_half_half_half<<<gridDim, blockDim>>>(hbsrVal, dB, dC, 
                                                      m, n, k, 
                                                      hf_alpha, hf_beta,
                                                      bsrRowPtr, bsrColInd);
   }
   bsrwmma_timer.Stop();
   double bsrwmma_time = bsrwmma_timer.ElapsedMillis() / double(TEST_TIMES);
   // ------
   hipMemcpy(hC, dC, m * n * sizeof(__half), hipMemcpyDeviceToHost);
   hipFree(dB);
   hipFree(dC);

   return bsrwmma_time;
}

bool verifyResult(__half *res1, __half *res2, int arrsize)
{
    // verify
    bool pass = true;
    for(int i=0; i<arrsize; i++)
    {
      if (static_cast<float>(res1[i]) != static_cast<float>(res2[i])) 
         {pass = false; break;}
    }
}

// blocked-ell storage
void BSR2BlockedELLhalf(int &ell_width, int *ell_columns, __half *ell_values,
                        __half *hA, int m, int k, 
                        int nblockrows, int nblocks, int block_dim,
                        int *bsrRowPtr, int *bsrColInd)
{
   // dense A info
   int   num_rows     = m;
   int   num_cols     = k;
   int   ld           = num_cols;
   int   dense_size   = ld * num_rows;
   __half *h_dense = hA;

   // bsr to host for conversion need
   int *h_bsrRowPtr = (int *) malloc(sizeof(int) * (nblockrows + 1));
   int *h_bsrColInd = (int *) malloc(sizeof(int) * nblocks);
   hipMemcpy(h_bsrRowPtr, bsrRowPtr, sizeof(int) * (nblockrows + 1), hipMemcpyDeviceToHost);
   hipMemcpy(h_bsrColInd, bsrColInd, sizeof(int) * nblocks, hipMemcpyDeviceToHost);

   int ell_blk_size = block_dim;
   ell_width = get_ell_width(h_bsrRowPtr, nblockrows) * ell_blk_size;
   int nnz = ell_width * num_rows;

   // set h_ell_columns
   int *h_ell_columns = (int*)malloc(sizeof(int) * nnz / (ell_blk_size * ell_blk_size));
   memset(h_ell_columns, 0, (nnz / (ell_blk_size * ell_blk_size)) * sizeof(int));
   fill_h_ell_columns(h_ell_columns, h_bsrRowPtr, h_bsrColInd, nblockrows, ell_width/ell_blk_size);
   free(h_bsrRowPtr);
   free(h_bsrColInd);

   // set empty h_ell_values
   __half* h_ell_values = (__half*)malloc(nnz * sizeof(__half));
   memset(h_ell_values, static_cast<__half>(0.0f), nnz*sizeof(__half));

   //--------------------------dense2sparse using cuSPARSE APIs--------------------------------
   // Device memory management
   int   *d_ell_columns;
   __half *d_ell_values,  *d_dense;
   hipMalloc((void**) &d_dense, dense_size * sizeof(__half));
   hipMalloc((void**) &d_ell_columns, nnz / (ell_blk_size * ell_blk_size) * sizeof(int));
   hipMalloc((void**) &d_ell_values, nnz * sizeof(__half));
   hipMemcpy(d_dense, h_dense, dense_size * sizeof(__half), hipMemcpyHostToDevice);
   hipMemcpy(d_ell_columns, h_ell_columns, 
            nnz / (ell_blk_size * ell_blk_size) * sizeof(int), 
            hipMemcpyHostToDevice);
   hipMemcpy(d_ell_values, h_ell_values, 
            nnz * sizeof(__half),
            hipMemcpyHostToDevice);

   // CUSPARSE APIs
   hipsparseHandle_t     handle = NULL;
   hipsparseSpMatDescr_t matB;
   hipsparseDnMatDescr_t matA;
   void*                dBuffer    = NULL;
   size_t               bufferSize = 0;
   hipsparseCreate(&handle);

   // Create dense matrix A
   hipsparseCreateDnMat(&matA, num_rows, num_cols, ld, d_dense,
                     HIP_R_16F, HIPSPARSE_ORDER_ROW);

   // Create sparse matrix B in Blocked ELL format
   hipsparseCreateBlockedEll(&matB, num_rows, num_cols,
                           ell_blk_size, ell_width,
                           d_ell_columns, d_ell_values,
                           HIPSPARSE_INDEX_32I,
                           HIPSPARSE_INDEX_BASE_ZERO,
                           HIP_R_16F);

   // allocate an external buffer if needed
   hipsparseDenseToSparse_bufferSize(handle, matA, matB,
                                 HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                 &bufferSize);
   hipMalloc(&dBuffer, bufferSize);

   // analyze Sparse to Dense conversion
   hipsparseDenseToSparse_analysis(handle, matA, matB,
                                 HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                 dBuffer);

   // execute Sparse to Dense conversion
   hipsparseDenseToSparse_convert(handle, matA, matB,
                              HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                              dBuffer);

   // destroy matrix/vector descriptors
   hipsparseDestroyDnMat(matA);
   hipsparseDestroySpMat(matB);
   hipsparseDestroy(handle);

   hipMemcpy(ell_columns, d_ell_columns, 
               nnz / (ell_blk_size * ell_blk_size) * sizeof(int), 
               hipMemcpyDeviceToDevice);
   hipMemcpy(ell_values, d_ell_values, 
               nnz * sizeof(__half),
               hipMemcpyDeviceToDevice);

   // free unused storage
   hipFree(dBuffer);
   hipFree(d_dense);
}